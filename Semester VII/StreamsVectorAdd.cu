#include "hip/hip_runtime.h"
// lab6.cu
// Completed by Jacob Doiron and Andrew Riehl
// COSC 480: GPU Programming
// 10/30/15
// Project which demonstrates the addition of two vectors, using multiple streams

// Important to note results will result in a loss of precision due to using
// floating point numbers instead of doubles (twice the precision, twice the memory).

#include <stdlib.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include "./common/book.h"

#define N   (1024 * 1024)
#define FULL_DATA_SIZE   (N * 20)

// Takes in three vectors a, b, and c, and adds vectors a and b into vector c
__global__ void vectorAdd(const float *a, const float *b, float *c)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < N) {
    float result = a[i] + b[i];
    c[i] = result;
  }
}

// Method which generates a random float value up to the specified max
float randomFloat(int max)
{
  int random = rand() % max;
  float scalar = (float) rand() / RAND_MAX;
  float result = random * scalar;
  return result;
}

// Main method which runs everything
int main(void)
{

  int whichDevice;
  float elapsed;

  hipDeviceProp_t prop;
  hipEvent_t start;
  hipEvent_t stop;
  hipStream_t stream0;
  hipStream_t stream1;
  hipStream_t stream2;
  hipStream_t stream3;

  // Create the timers
  HANDLE_ERROR(hipEventCreate(&start));
  HANDLE_ERROR(hipEventCreate(&stop));

  // Initialize the streams
  HANDLE_ERROR(hipStreamCreate(&stream0));
  HANDLE_ERROR(hipStreamCreate(&stream1));
  HANDLE_ERROR(hipStreamCreate(&stream2));
  HANDLE_ERROR(hipStreamCreate(&stream3));

  float *host_a, *host_b, *host_c;

  float *device_a0, *device_b0, *device_c0;
  float *device_a1, *device_b1, *device_c1;
  float *device_a2, *device_b2, *device_c2;
  float *device_a3, *device_b3, *device_c3;

  HANDLE_ERROR(hipGetDevice(&whichDevice));
  HANDLE_ERROR(hipGetDeviceProperties(&prop, whichDevice));

  if (!prop.deviceOverlap) {
    printf("Device will not handle overlaps, so no speed up from streams\n");
    return 0;
  }

  // Seed the random function to get random values from #rand
  srand(time(NULL));

  printf("Performing test on vectors, each with size %d (elements)\n", FULL_DATA_SIZE);

  // Allocate GPU memory
  HANDLE_ERROR(hipMalloc((void**) &device_a0, N * sizeof(float)));
  HANDLE_ERROR(hipMalloc((void**) &device_b0, N * sizeof(float)));
  HANDLE_ERROR(hipMalloc((void**) &device_c0, N * sizeof(float)));

  HANDLE_ERROR(hipMalloc((void**) &device_a1, N * sizeof(float)));
  HANDLE_ERROR(hipMalloc((void**) &device_b1, N * sizeof(float)));
  HANDLE_ERROR(hipMalloc((void**) &device_c1, N * sizeof(float)));

  HANDLE_ERROR(hipMalloc((void**) &device_a2, N * sizeof(float)));
  HANDLE_ERROR(hipMalloc((void**) &device_b2, N * sizeof(float)));
  HANDLE_ERROR(hipMalloc((void**) &device_c2, N * sizeof(float)));

  HANDLE_ERROR(hipMalloc((void**) &device_a3, N * sizeof(float)));
  HANDLE_ERROR(hipMalloc((void**) &device_b3, N * sizeof(float)));
  HANDLE_ERROR(hipMalloc((void**) &device_c3, N * sizeof(float)));

  // Allocate host locked memory for streams
  HANDLE_ERROR(hipHostAlloc((void**) &host_a, FULL_DATA_SIZE * sizeof(float), hipHostMallocDefault));
  HANDLE_ERROR(hipHostAlloc((void**) &host_b, FULL_DATA_SIZE * sizeof(float), hipHostMallocDefault));
  HANDLE_ERROR(hipHostAlloc((void**) &host_c, FULL_DATA_SIZE * sizeof(float), hipHostMallocDefault));

  // Populate host vectors
  for (int i = 0; i < FULL_DATA_SIZE; i++) {
    host_a[i] = randomFloat(100);
    host_b[i] = randomFloat(100);
  }

  // Record the start time
  HANDLE_ERROR(hipEventRecord(start, 0));

  for (int i = 0; i < FULL_DATA_SIZE; i += N * 4) {
    // Enqueue a into the streams
    HANDLE_ERROR(hipMemcpyAsync(device_a0, host_a + i, N * sizeof(float), hipMemcpyHostToDevice, stream0));
    HANDLE_ERROR(hipMemcpyAsync(device_a1, host_a + i, N * sizeof(float), hipMemcpyHostToDevice, stream1));
    HANDLE_ERROR(hipMemcpyAsync(device_a2, host_a + i, N * sizeof(float), hipMemcpyHostToDevice, stream2));
    HANDLE_ERROR(hipMemcpyAsync(device_a3, host_a + i, N * sizeof(float), hipMemcpyHostToDevice, stream3));

    // Enqueue b into the streams
    HANDLE_ERROR(hipMemcpyAsync(device_b0, host_b + i, N * sizeof(float), hipMemcpyHostToDevice, stream0));
    HANDLE_ERROR(hipMemcpyAsync(device_b1, host_b + i, N * sizeof(float), hipMemcpyHostToDevice, stream1));
    HANDLE_ERROR(hipMemcpyAsync(device_b2, host_b + i, N * sizeof(float), hipMemcpyHostToDevice, stream2));
    HANDLE_ERROR(hipMemcpyAsync(device_b3, host_b + i, N * sizeof(float), hipMemcpyHostToDevice, stream3));

    // Enqueue the kernel into the streams
    vectorAdd<<<N / 256, 256, 0, stream0>>>(device_a0, device_b0, device_c0);
    vectorAdd<<<N / 256, 256, 0, stream1>>>(device_a1, device_b1, device_c1);
    vectorAdd<<<N / 256, 256, 0, stream2>>>(device_a2, device_b2, device_c2);
    vectorAdd<<<N / 256, 256, 0, stream3>>>(device_a3, device_b3, device_c3);

    // Enqueue c from device to locked memory
    HANDLE_ERROR(hipMemcpyAsync(host_c + i, device_c0, N * sizeof(float), hipMemcpyDeviceToHost, stream0));
    HANDLE_ERROR(hipMemcpyAsync(host_c + i + N, device_c1, N * sizeof(float), hipMemcpyDeviceToHost, stream1));
    HANDLE_ERROR(hipMemcpyAsync(host_c + i + (N * 2), device_c2, N * sizeof(float), hipMemcpyDeviceToHost, stream2));
    HANDLE_ERROR(hipMemcpyAsync(host_c + i + (N * 3), device_c3, N * sizeof(float), hipMemcpyDeviceToHost, stream3));
  }

  // Synchronizes all streams
  HANDLE_ERROR(hipStreamSynchronize(stream0));
  HANDLE_ERROR(hipStreamSynchronize(stream1));
  HANDLE_ERROR(hipStreamSynchronize(stream2));
  HANDLE_ERROR(hipStreamSynchronize(stream3));

  // Record the stop time
  HANDLE_ERROR(hipEventRecord(stop, 0));

  // Synchronize the stop timer
  HANDLE_ERROR(hipEventSynchronize(stop));

  // Record the elapsed time between the timers
  HANDLE_ERROR(hipEventElapsedTime(&elapsed, start, stop));

  printf("Time taken: %3.1f ms\nVALIDATION:\n", elapsed);

  // Test a small subset of the results
  for (int i = 0; i < 10; i++) {
    printf("a[%d]: %9.6f, b[%d]: %9.6f, c[%d]: %9.6f\n", i, host_a[i], i, host_b[i], i, host_c[i]);
  }

  // Frees all device memory
  HANDLE_ERROR(hipFree(device_a0));
  HANDLE_ERROR(hipFree(device_b0));
  HANDLE_ERROR(hipFree(device_c0));

  HANDLE_ERROR(hipFree(device_a1));
  HANDLE_ERROR(hipFree(device_b1));
  HANDLE_ERROR(hipFree(device_c1));

  HANDLE_ERROR(hipFree(device_a2));
  HANDLE_ERROR(hipFree(device_b2));
  HANDLE_ERROR(hipFree(device_c2));

  HANDLE_ERROR(hipFree(device_a3));
  HANDLE_ERROR(hipFree(device_b3));
  HANDLE_ERROR(hipFree(device_c3));

  printf("All device memory has been freed.\n");

  // Frees all host memory
  HANDLE_ERROR(hipHostFree(host_a));
  HANDLE_ERROR(hipHostFree(host_b));
  HANDLE_ERROR(hipHostFree(host_c));

  printf("All host memory has been freed.\n");

  // Destroys all streams
  HANDLE_ERROR(hipStreamDestroy(stream0));
  HANDLE_ERROR(hipStreamDestroy(stream1));
  HANDLE_ERROR(hipStreamDestroy(stream2));
  HANDLE_ERROR(hipStreamDestroy(stream3));

  printf("All streams have been destroyed.\n");

  // Destroys timer events
  HANDLE_ERROR(hipEventDestroy(start));
  HANDLE_ERROR(hipEventDestroy(stop));

  printf("All timers have been destroyed.\n");

  return 0;
}
