#include "hip/hip_runtime.h"
#include "./common/book.h"
#include "./common/cpu_bitmap.h"

#include <time.h>

#define WIDTH 800
#define HEIGHT 608

/**
 * @author Jacob Doiron
 * @since 12/6/15
 */

struct Complex {

  float real;
  float imaginary;

  Complex(const float r, const float i) {
    real = r;
    imaginary = i;
  }

  float magnitude2(void) {
    return (real * real) + (imaginary * imaginary);
  }

  Complex operator *(const Complex& a) {
    return Complex((real * a.real) - (imaginary * a.imaginary), (imaginary * a.real) + (real * a.imaginary));
  }

  Complex operator +(const Complex& a) {
    return Complex(real + a.real, imaginary + a.imaginary);
  }
};

void kernel(unsigned char *buffer) {
  for (int column = 0; column < WIDTH; column++) {
    for (int row = 0; row < HEIGHT; row++) {
      const int offset = (row * WIDTH) + column;
      const float x0 = (((float) column / WIDTH) * 3.5F) - 2.5F;
      const float y0 = (((float) row / HEIGHT) * 3.5F) - 1.75F;
      float x = 0.0F;
      float y = 0.0F;
      float temporary_x;
      int i;
      for (i = 0; i < 100 && (x * x) + (y * y) <= 4.0F; i++) {
        temporary_x = (x * x) - (y * y) + x0;
        y = (2.0F * x * y) + y0;
        x = temporary_x;
      }
      int color = i * 5;
      if (color >= 256) {
        color = 0;
      }
      const int index = offset * 4;
      buffer[index] = 0;
      buffer[index + 1] = color;
      buffer[index + 2] = 0;
      buffer[index + 3] = 255;
    }
  }
}

int main(void) {
  CPUBitmap bitmap(WIDTH, HEIGHT);
  time_t start = time(0);
  unsigned char *buffer = bitmap.get_ptr();
  kernel(buffer);
  time_t end = time(0);
  printf("Mandelbrot fractal created in %03.0f secs\n", difftime(end, start));
  bitmap.display_and_exit();
}
