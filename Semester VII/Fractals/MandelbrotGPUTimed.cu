#include "hip/hip_runtime.h"
#include "./common/book.h"
#include "./common/cpu_bitmap.h"

#define WIDTH 800
#define HEIGHT 608

/**
 * @author Jacob Doiron
 * @since 12/6/15
 */

__global__ void kernel(unsigned char *buffer) {
  const int row = (blockIdx.y * blockDim.y) + threadIdx.y;
  const int column = (blockIdx.x * blockDim.x) + threadIdx.x;
  const int offset = (row * gridDim.x * blockDim.x) + column;
  if(column >= WIDTH || row >= HEIGHT) {
    return;
  }
  const float x0 = (((float) column / WIDTH) * 3.5F) - 2.5F;
  const float y0 = (((float) row / HEIGHT) * 3.5F) - 1.75F;
  float x = 0.0F;
  float y = 0.0F;
  float temporary_x;
  int i;
  for (i = 0; i < 100 && (x * x) + (y * y) <= 4.0F; i++) {
    temporary_x = (x * x) - (y * y) + x0;
    y = (2.0F * x * y) + y0;
    x = temporary_x;
  }
  int color = i * 5;
  if (color >= 256) {
    color = 0;
  }
  const int index = offset * 4;
  buffer[index] = 0;
  buffer[index + 1] = color;
  buffer[index + 2] = 0;
  buffer[index + 3] = 255;
}

int main(void) {
  CPUBitmap bitmap(WIDTH, HEIGHT);
  unsigned char *dev_bitmap;
  float elapsed;

  dim3 block_size(16, 16);
  dim3 grid_size(WIDTH / block_size.x, HEIGHT / block_size.y);

  hipEvent_t start, stop;
  hipEvent_t bitmapCpy_start, bitmapCpy_stop;

  HANDLE_ERROR(hipEventCreate(&start));
  HANDLE_ERROR(hipEventCreate(&stop));
  HANDLE_ERROR(hipEventCreate(&bitmapCpy_start));
  HANDLE_ERROR(hipEventCreate(&bitmapCpy_stop));

  HANDLE_ERROR(hipMalloc((void**) &dev_bitmap, bitmap.image_size()));
  HANDLE_ERROR(hipEventRecord(start, 0));

  for (int i = 0; i < 1000; i++) {
    kernel<<<grid_size, block_size>>>(dev_bitmap);
  }

  HANDLE_ERROR(hipMemcpy(bitmap.get_ptr(), dev_bitmap, bitmap.image_size(), hipMemcpyDeviceToHost));

  HANDLE_ERROR(hipEventRecord(stop, 0));
  HANDLE_ERROR(hipEventSynchronize(stop));
  HANDLE_ERROR(hipEventElapsedTime(&elapsed, start, stop));

  printf("Mandelbrot fractal created 1000x in %3.1f ms\n", elapsed);

  bitmap.display_and_exit();

  HANDLE_ERROR(hipEventDestroy(start));
  HANDLE_ERROR(hipEventDestroy(stop));
  HANDLE_ERROR(hipEventDestroy(bitmapCpy_start));
  HANDLE_ERROR(hipEventDestroy(bitmapCpy_stop));
  HANDLE_ERROR(hipFree(dev_bitmap));
}
