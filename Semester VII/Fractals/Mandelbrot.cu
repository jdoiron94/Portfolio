#include "hip/hip_runtime.h"
#include "./common/book.h"
#include "./common/cpu_bitmap.h"

#define WIDTH 800
#define HEIGHT 608

__global__ void kernel(char *buffer) {
	const int row = (blockIdx.y * blockDim.y) + threadIdx.y;
	const int column = (blockIdx.x * blockDim.x) + threadIdx.x;
	const int offset = (row * gridDim.x * blockDim.x) + column;
	if(column >= WIDTH || row >= HEIGHT) {
		return;
	}
	const float x0 = (((float) column / WIDTH) * 3.5F) - 2.5F;
	const float y0 = (((float) row / HEIGHT) * 3.5F) - 1.75F;
	float x = 0.0F;
	float y = 0.0F;
	float temporary_x;
	int i;
	for (i = 0; i < 100 && (x * x) + (y * y) <= 4.0F; i++) { 
		temporary_x = (x * x) - (y * y) + x0;
		y = (2.0F * x * y) + y0;
		x = temporary_x;
	}
	int color = i * 5;
	if (color >= 256) {
		color = 0;
	}
	buffer[(offset * 4) + 1] = color;
}

int main(void) {
	CPUBitmap bitmap(WIDTH, HEIGHT);
	char *dev_bitmap;
	float elapsed;
	
	dim3 block_size(16, 16);
	dim3 grid_size(WIDTH / block_size.x, HEIGHT / block_size.y);

	hipEvent_t start, stop;
	hipEvent_t bitmapCpy_start, bitmapCpy_stop;

	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipEventCreate(&bitmapCpy_start));
	HANDLE_ERROR(hipEventCreate(&bitmapCpy_stop));

	HANDLE_ERROR(hipMalloc((void**) &dev_bitmap, bitmap.image_size()));
	HANDLE_ERROR(hipEventRecord(start, 0));

	kernel<<<grid_size, block_size>>>(dev_bitmap);

	HANDLE_ERROR(hipMemcpy(bitmap.get_ptr(), dev_bitmap, bitmap.image_size(), hipMemcpyDeviceToHost));

	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	HANDLE_ERROR(hipEventElapsedTime(&elapsed, start, stop));

	printf("Mandelbrot fractal created and copied back to host memory in %3.1f ms\n", elapsed);

	bitmap.display_and_exit();

	HANDLE_ERROR(hipEventDestroy(start));
	HANDLE_ERROR(hipEventDestroy(stop));
	HANDLE_ERROR(hipEventDestroy(bitmapCpy_start));
	HANDLE_ERROR(hipEventDestroy(bitmapCpy_stop));
	HANDLE_ERROR(hipFree(dev_bitmap));
}