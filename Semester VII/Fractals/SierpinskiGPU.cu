#include "hip/hip_runtime.h"
#include "./common/book.h"
#include "./common/cpu_bitmap.h"

#define WIDTH 800
#define HEIGHT 608

/**
 * This method takes an x and y coordinate and subdivides up the values.
 * It checks to see if the coorinate is at the top left edge of the square
 * and will then subdivide the square into nine, removing the centermost.
 * It then does the same to each of the other eight squares
 */
__device__ int sierpinski(int x, int y) {
	for (;x > 0 || y > 0;) {
		if (x % 3 == 1 && y % 3 == 1) {
			return 0;
		}
		x /= 3;
		y /= 3;
	}
	return 1;
}

/**
 * Runs our GPU kernel
 */
__global__ void kernel(unsigned char *buffer) {
	const int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	const int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	const int offset = (y * gridDim.x * blockDim.x) + x;
	const int sier = sierpinski(x, y);
	const int index = offset * 4;
	buffer[index] = 0;
	buffer[index + 1] = (x * 256) / 800 * sier;
	buffer[index + 2] = (y * 256) / 608 * sier;
	buffer[index + 3] = 255;
}

/**
 * Main method to time our GPU kernel 1000x and display the bitmap
 */
int main(void) {
	CPUBitmap bitmap(WIDTH, HEIGHT);
	unsigned char *dev_bitmap;
	float elapsed;

	dim3 block_size(16, 16);
	dim3 grid_size(WIDTH / block_size.x, HEIGHT / block_size.y);

	hipEvent_t start, stop;
	hipEvent_t bitmapCpy_start, bitmapCpy_stop;

	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipEventCreate(&bitmapCpy_start));
	HANDLE_ERROR(hipEventCreate(&bitmapCpy_stop));

	HANDLE_ERROR(hipMalloc((void**) &dev_bitmap, bitmap.image_size()));

	HANDLE_ERROR(hipEventRecord(start, 0));

	for (int i = 0; i < 1000; i++) {
		kernel<<<grid_size, block_size>>>(dev_bitmap);
	}

	HANDLE_ERROR(hipMemcpy(bitmap.get_ptr(), dev_bitmap, bitmap.image_size(), hipMemcpyDeviceToHost));

	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));

	HANDLE_ERROR(hipEventElapsedTime(&elapsed, start, stop));

	printf("Sierpinski carpet fractal created 1000x and copied back to host memory in %3.1f ms\n", elapsed);

	bitmap.display_and_exit();

	HANDLE_ERROR(hipEventDestroy(start));
	HANDLE_ERROR(hipEventDestroy(stop));
	HANDLE_ERROR(hipEventDestroy(bitmapCpy_start));
	HANDLE_ERROR(hipEventDestroy(bitmapCpy_stop));
	HANDLE_ERROR(hipFree(dev_bitmap));
}