#include "hip/hip_runtime.h"
#include "./common/book.h"
#include "./common/cpu_bitmap.h"

#include <stdio.h>
#include <time.h>

#define DIM 1024

struct hipComplex {

	float r;
	float i;

	__device__ hipComplex(const float a, const float b) {
		r = a;
		i = b;
	}

	__device__ float magnitude2(void) {
		return (r * r) + (i * i);
	}

	__device__ hipComplex operator *(const hipComplex& a) {
		return hipComplex((r * a.r) - (i * a.i), (i * a.r) + (r * a.i));
	}

	__device__ hipComplex operator +(const hipComplex& a) {
		return hipComplex(r + a.r, i + a.i);
	}
};

__device__ int julia(const int x, const int y) {
	const float scale = 1.5F;
	const float div_dim = (float) DIM / 2;
	const float julia_x = scale * (div_dim - x) / div_dim;
	const float julia_y = scale * (div_dim - y) / div_dim;
	
	//hipComplex c(-0.8F, 0.156F);
	hipComplex c(-0.8F, 0.15F);
	hipComplex a(julia_x, julia_y);

	for (int i = 0; i < 200; i++) {
		a = (a * a) + c;
		if (a.magnitude2() > 2) {
			return 0;
		}
	}
	return 1;
}

__global__ void kernel(unsigned char *ptr, const int red, const int green, const int blue) {
	const int x = threadIdx.x + (blockIdx.x * blockDim.x);
	const int y = threadIdx.y + (blockIdx.y * blockDim.y);
	const int offset = x + ((y * gridDim.x) * blockDim.x);
	const int juliaValue = julia(x, y);
	int index = offset * 4;
	ptr[index++] = red * juliaValue;
	ptr[index++] = green * juliaValue;
	ptr[index++] = blue * juliaValue;
	ptr[index] = 0;
}

int main(void)
{

	CPUBitmap bitmap(DIM, DIM);

	unsigned char * dev_bitmap;
	float elapsed;

	dim3 grid(DIM, DIM);
	dim3 blocks(DIM / 16, DIM / 16);
	dim3 threads(16, 16);
	dim3 thread(DIM, DIM);

	hipEvent_t start, stop;
	hipEvent_t bitmapCpy_start, bitmapCpy_stop;

	srand(time(NULL));

	const int red = rand() % 255;
	const int green = rand() % 255;
	const int blue = rand() % 255;

	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipEventCreate(&bitmapCpy_start));
	HANDLE_ERROR(hipEventCreate(&bitmapCpy_stop));

	HANDLE_ERROR(hipMalloc((void**) &dev_bitmap, bitmap.image_size()));

	HANDLE_ERROR(hipEventRecord(start, 0));

	kernel << <blocks, threads >> >(dev_bitmap, red, green, blue);

	HANDLE_ERROR(hipMemcpy(bitmap.get_ptr(), dev_bitmap, bitmap.image_size(), hipMemcpyDeviceToHost));

	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));

	HANDLE_ERROR(hipEventElapsedTime(&elapsed, start, stop));

	printf("Julia fractal created and copied back to host memory in %3.1f ms\n", elapsed);

	bitmap.display_and_exit();

	HANDLE_ERROR(hipEventDestroy(start));
	HANDLE_ERROR(hipEventDestroy(stop));
	HANDLE_ERROR(hipEventDestroy(bitmapCpy_start));
	HANDLE_ERROR(hipEventDestroy(bitmapCpy_stop));
	HANDLE_ERROR(hipFree(dev_bitmap));
}