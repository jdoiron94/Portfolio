#include "hip/hip_runtime.h"
#include "./common/book.h"
#include "./common/cpu_bitmap.h"

#define WIDTH 800
#define HEIGHT 608

struct hipComplex {

	float r;
	float i;

	__device__ hipComplex(const float a, const float b) {
		r = a;
		i = b;
	}

	__device__ float magnitude2(void) {
		return (r * r) + (i * i);
	}

	__device__ hipComplex operator *(const hipComplex& a) {
		return hipComplex((r * a.r) - (i * a.i), (i * a.r) + (r * a.i));
	}

	__device__ hipComplex operator +(const hipComplex& a) {
		return hipComplex(r + a.r, i + a.i);
	}
};

__device__ int julia(const int x, const int y) {
	const float column = (((float) x / WIDTH) * 3.5F) - 1.75F;
	const float row = (((float) y / HEIGHT) * 3.5F) - 1.75F;
	
	hipComplex c(-0.8F, 0.15F);
	hipComplex a(column, row);

	for (int i = 0; i < 200; i++) {
		a = (a * a) + c;
		if (a.magnitude2() > 2) {
			return 0;
		}
	}
	return 1;
}

__global__ void kernel(char *buffer) {
	const int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	const int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	const int offset = (y * gridDim.x * blockDim.x) + x;
	const int juliaValue = julia(x, y);
	const int index = offset * 4;
	buffer[index + 1] = (x * 256) / 800 * juliaValue;
	buffer[index + 2] = (y * 256) / 608 * juliaValue;
}

int main(void) {
	CPUBitmap bitmap(WIDTH, HEIGHT);
	char *dev_bitmap;
	float elapsed;

	dim3 block_size(16, 16);
	dim3 grid_size(WIDTH / block_size.x, HEIGHT / block_size.y);

	hipEvent_t start, stop;
	hipEvent_t bitmapCpy_start, bitmapCpy_stop;

	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipEventCreate(&bitmapCpy_start));
	HANDLE_ERROR(hipEventCreate(&bitmapCpy_stop));

	HANDLE_ERROR(hipMalloc((void**) &dev_bitmap, bitmap.image_size()));

	HANDLE_ERROR(hipEventRecord(start, 0));

	kernel<<<grid_size, block_size>>>(dev_bitmap);

	HANDLE_ERROR(hipMemcpy(bitmap.get_ptr(), dev_bitmap, bitmap.image_size(), hipMemcpyDeviceToHost));

	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));

	HANDLE_ERROR(hipEventElapsedTime(&elapsed, start, stop));

	printf("Julia fractal created and copied back to host memory in %3.1f ms\n", elapsed);

	bitmap.display_and_exit();

	HANDLE_ERROR(hipEventDestroy(start));
	HANDLE_ERROR(hipEventDestroy(stop));
	HANDLE_ERROR(hipEventDestroy(bitmapCpy_start));
	HANDLE_ERROR(hipEventDestroy(bitmapCpy_stop));
	HANDLE_ERROR(hipFree(dev_bitmap));
}