#include "hip/hip_runtime.h"
#include "./common/book.h"
#include "./common/cpu_bitmap.h"

#define WIDTH 800
#define HEIGHT 608

struct Complex {

	float real;
	float imaginary;

	__device__ Complex(const float r, const float i) {
		real = r;
		imaginary = i;
	}

	__device__ float magnitude2(void) {
		return (real * real) + (imaginary * imaginary);
	}

	__device__ Complex operator *(const Complex& a) {
		return Complex((real * a.real) - (imaginary * a.imaginary), (imaginary * a.real) + (real * a.imaginary));
	}

	__device__ Complex operator +(const Complex& a) {
		return Complex(real + a.real, imaginary + a.imaginary);
	}
};

__device__ int julia(const int x, const int y) {
	const float column = (((float) x / WIDTH) * 3.5F) - 1.75F;
	const float row = (((float) y / HEIGHT) * 3.5F) - 1.75F;
	
	struct Complex c(-0.8F, 0.15F);
	struct Complex a(column, row);

	for (int i = 0; i < 200; i++) {
		a = (a * a) + c;
		if (a.magnitude2() > 1000) {
			return 0;
		}
	}
	return 1;
}

__global__ void kernel(unsigned char *buffer) {
	const int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	const int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	const int offset = (y * gridDim.x * blockDim.x) + x;
	const int juliaValue = julia(x, y);
	const int index = offset * 4;
	buffer[index] = 0;
	buffer[index + 1] = (x * 256) / 800 * juliaValue;
	buffer[index + 2] = (y * 256) / 608 * juliaValue;
}

int main(void) {
	CPUBitmap bitmap(WIDTH, HEIGHT);
	unsigned char *dev_bitmap;
	float elapsed;

	dim3 block_size(16, 16);
	dim3 grid_size(WIDTH / block_size.x, HEIGHT / block_size.y);

	hipEvent_t start, stop;
	hipEvent_t bitmapCpy_start, bitmapCpy_stop;

	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipEventCreate(&bitmapCpy_start));
	HANDLE_ERROR(hipEventCreate(&bitmapCpy_stop));

	HANDLE_ERROR(hipMalloc((void**) &dev_bitmap, bitmap.image_size()));

	HANDLE_ERROR(hipEventRecord(start, 0));

	for (int i = 0; i < 1000; i++) {
		kernel<<<grid_size, block_size>>>(dev_bitmap);
	}

	HANDLE_ERROR(hipMemcpy(bitmap.get_ptr(), dev_bitmap, bitmap.image_size(), hipMemcpyDeviceToHost));

	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));

	HANDLE_ERROR(hipEventElapsedTime(&elapsed, start, stop));

	printf("Julia fractal created 1000x in %05.1f ms\n", elapsed);

	bitmap.display_and_exit();

	HANDLE_ERROR(hipEventDestroy(start));
	HANDLE_ERROR(hipEventDestroy(stop));
	HANDLE_ERROR(hipEventDestroy(bitmapCpy_start));
	HANDLE_ERROR(hipEventDestroy(bitmapCpy_stop));
	HANDLE_ERROR(hipFree(dev_bitmap));
}