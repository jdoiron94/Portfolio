#include "hip/hip_runtime.h"
// proj2.cu
// Merle Ferguson and Jacob Doiron
// COSC 480: GPU Programming
// 10/16/15
// Project which demonstrates parallel matrix addition, using shared memory


// We experienced the atomicAdd kernel taking longer (about twice as long)
// as the regular add. We believe this is because, despite atomic additions
// being very quick, only one happens at a time. Performing a bunch of
// slow adds at the same time in our case was quicker than performing
// a bunch of quick adds sequentially.

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <hip/hip_runtime.h>

#define threadsPerBlock 256

// Accepts a 2D array as a parameter and flattens it into a 1D array
void flattenHostMatrix(int **h_a, int *r_a, const int rows, const int columns)
{
  for (int i = 0; i < rows; i++) {
    for (int j = 0; j < columns; j++) {
      r_a[j + i * columns] = h_a[i][j];
    }
  }
}

// Accepts a 1D array as a parameter and expands it into a 2D array
void expandHostMatrix(int **h_c, const int *r_c, const int rows, const int columns)
{
  for (int i = 0; i < rows; i++) {
    for (int j = 0; j < columns; j++) {
      h_c[i][j] = r_c[j + i * columns];
    }
  }
}

// Mallocs the 2D host array and returns it
int **createHostMatrix(const int rows, const int columns)
{
  int **host = (int**) malloc(rows * sizeof(int*));
  if (host == NULL) {
    printf("Failed to allocate host matrix!\n");
    exit(EXIT_FAILURE);
  }
  for (int i = 0; i < rows; i++) {
    host[i] = (int*) malloc(columns * sizeof(int));
    if (host[i] == NULL) {
      printf("Failed to allocate host matrix!\n");
      exit(EXIT_FAILURE);
    }
  }
  return host;
}

// Fills the random values of the 2D host arrays
int **initializeHostMatrix(const int rows, const int columns)
{
  int **host = createHostMatrix(rows, columns);
  for (int i = 0; i < rows; i++) {
    for (int j = 0; j < columns; j++) {
      host[i][j] = rand() % 100;
    }
  }
  return host;
}

// Frees the 2D host arrays
void freeHostMatrix(int **host, const int rows)
{
  for (int i = 0; i < rows; i++) {
    free(host[i]);
  }
  free(host);
}

// Mallocs the device arrays
int *createDeviceMatrix(hipError_t err, const int rows, const int columns)
{
  int *device = NULL;
  err = hipMalloc((void**) &device, rows * columns * sizeof(int));
  if (err != hipSuccess) {
    printf("Failed to allocate device matrix (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  return device;
}

// Initializes the device array with the provided (flattened) host array
int *initializeDeviceMatrix(hipError_t err, const int *host, const int rows, const int columns)
{
  int *device = createDeviceMatrix(err, rows, columns);
  err = hipMemcpy(device, host, rows * columns * sizeof(int), hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    printf("Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  return device;
}

// Frees the device arrays
void freeDeviceMatrix(hipError_t err, int *device)
{
  err = hipFree(device);
  if (err != hipSuccess)  {
    printf("Failed to free device matrix (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

// Sums two matrices in 1D array form and stores the result in shared memory.
// Then copies this result from shared memory to the array that is used later
__global__ void addMatrices(const int *a, const int *b, int *c, const int n, const int m)
{
  __shared__ int cache[threadsPerBlock];

  const unsigned int i = threadIdx.x;
  unsigned int tid = blockDim.x * blockIdx.x + i;
  const unsigned int stride = blockDim.x * gridDim.x;

  while (tid < n * m) {
    // Store the sum for all threads in the block
    cache[i] = a[tid] + b[tid];
    __syncthreads();
    // Copy the sums from the cache to the return array
    c[tid] = cache[i];
    __syncthreads();
    // Skip the total number of threads in the grid so this thread can work on a new unique element
    tid += stride;
  }
}

// Sums two matrices in 1d array form and stores the result in shared memory. Then copies this result from shared memory to the array that is used later
// In particular, this uses the atomicAdd for additions
__global__ void addMatricesAtomically(const int *a, const int *b, int *c, const int n, const int m)
{
  __shared__ int cache[threadsPerBlock];

  const unsigned int i = threadIdx.x;
  unsigned int tid = blockDim.x * blockIdx.x + i;
  const unsigned int stride = blockDim.x * gridDim.x;

  while (tid < n * m) {
    // Store the sum for all threads in the block
    atomicAdd(&cache[i], a[i]);
    atomicAdd(&cache[i], b[i]);
    // Copy the sums from the cache to the return array
    c[tid] = cache[i];
    __syncthreads();
    // Skip the total number of threads in the grid so this thread can work on a new unique element
    tid += stride;
  }
}

// Calls functions to sum matrices in two different ways
void callKernel(hipError_t err, const int *device_a, const int *device_b, int *device_c, int *host_c, const int rows, const int columns, const int atomic)
{
  int blocksPerGrid = ((rows * columns) + threadsPerBlock - 1) / threadsPerBlock;
  if (atomic) {
    addMatricesAtomically<<<blocksPerGrid, threadsPerBlock>>>(device_a, device_b, device_c, rows, columns);
  } else {
    addMatrices<<<blocksPerGrid, threadsPerBlock>>>(device_a, device_b, device_c, rows, columns);
  }
  err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Failed to launch addMatrices kernel (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  err = hipMemcpy(host_c, device_c, (rows * columns*sizeof(int)), hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    printf("Failed to copy result matrix from device to host (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

// Calls functions to sum matrices in two different ways, recording the time each takes
int main()
{
  int rows;
  int columns;
  float elapsedTime;

  hipEvent_t start;
  hipEvent_t stop;
  hipError_t err = hipSuccess;

  printf("Please enter the number of rows per matrix: ");
  scanf("%d", &rows);
  printf("Please enter the number of columns per matrix: ");
  scanf("%d", &columns);

  srand(time(NULL));

  int **host_a = initializeHostMatrix(rows, columns);
  int **host_b = initializeHostMatrix(rows, columns);
  int **host_c = createHostMatrix(rows, columns);

  int *host_a_flat = (int*) malloc(rows * columns * sizeof(int));
  int *host_b_flat = (int*) malloc(rows * columns * sizeof(int));
  int *host_c_flat = (int*) malloc(rows * columns * sizeof(int));

  // Convert 2D matrices to 1D arrays so they can be copied into device memory
  flattenHostMatrix(host_a, host_a_flat, rows, columns);
  flattenHostMatrix(host_b, host_b_flat, rows, columns);

  int *device_a = initializeDeviceMatrix(err, host_a_flat, rows, columns);
  int *device_b = initializeDeviceMatrix(err, host_b_flat, rows, columns);
  int *device_c = createDeviceMatrix(err, rows, columns);

  err = hipEventCreate(&start);
  if (err != hipSuccess) {
    printf("Failed to create cuda event start (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  err = hipEventCreate(&stop);
  if (err != hipSuccess) {
    printf("Failed to create cuda event stop (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  err = hipEventRecord(start, 0);
  if (err != hipSuccess) {
    printf("Failed to record event start (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  callKernel(err, device_a, device_b, device_c, host_c_flat, rows, columns, 1);
  err = hipEventRecord(stop, 0);
  if (err != hipSuccess) {
    printf("Failed to record event stop (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  err = hipEventSynchronize(stop);
  if (err != hipSuccess) {
    printf("Failed to synchronize (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  err = hipEventElapsedTime(&elapsedTime, start, stop);
  if (err != hipSuccess) {
    printf("Failed to calculate elapsed time (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  printf("\nMatrix add for atomic %3.1f ms \n", elapsedTime);

  expandHostMatrix(host_c, host_c_flat, rows, columns);

  err = hipEventCreate(&start);
  if (err != hipSuccess) {
    printf("Failed to create cuda event start (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  err = hipEventCreate(&stop);
  if (err != hipSuccess) {
    printf("Failed to create cuda event stop (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  err = hipEventRecord(start, 0);
  if (err != hipSuccess) {
    printf("Failed to record event start (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  callKernel(err, device_a, device_b, device_c, host_c_flat, rows, columns, 0);
  err = hipEventRecord(stop, 0);
  if (err != hipSuccess) {
    printf("Failed to record event stop (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  err = hipEventSynchronize(stop);
  if (err != hipSuccess) {
    printf("Failed to synchronize (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  err = hipEventElapsedTime(&elapsedTime, start, stop);
  if (err != hipSuccess) {
    printf("Failed to calculate elapsed time (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  printf("\nMatrix add for non-atomic %3.1f ms \n", elapsedTime);

  expandHostMatrix(host_c, host_c_flat, rows, columns);

  printf("\nWriting results to results.txt . . .\n");

  // File pointer to record results
  FILE *file = fopen("results.txt", "w");

  if (file == NULL) {
    printf("Error opening file!\n");
    exit(1);
  }

  // Write original matrix to file
  fprintf(file, "Matrix A:\n");
  for (int i = 0; i < rows; i++) {
    for (int j = 0; j < columns; j++) {
      fprintf(file, "%3d ", host_a[i][j]);
    }
    fprintf(file, "\n");
  }

  fprintf(file, "\n\n");

  // Write original matrix to file
  fprintf(file, "Matrix B:\n");
  for (int i = 0; i < rows; i++) {
    for (int j = 0; j < columns; j++) {
      fprintf(file, "%3d ", host_b[i][j]);
    }
    fprintf(file, "\n");
  }

  fprintf(file, "\n\n");

  // Write resulting matrix to file
  fprintf(file, "Matrix Result:\n");
  for (int i = 0; i < rows; i++) {
    for (int j = 0; j < columns; j++) {
      fprintf(file, "%3d ", host_c[i][j]);
    }
    fprintf(file, "\n");
  }

  fprintf(file, "\n\n");
  fclose(file);

  printf("\nresults.txt is ready\n");

  freeDeviceMatrix(err, device_a);
  freeDeviceMatrix(err, device_b);
  freeDeviceMatrix(err, device_c);

  printf("\nDevice matrices freed\n");

  freeHostMatrix(host_a);
  freeHostMatrix(host_b);
  freeHostMatrix(host_c);

  free(host_a_flat);
  free(host_b_flat);
  free(host_c_flat);

  printf("\nHost matrices freed\n");

  err = hipDeviceReset();
  if (err != hipSuccess)  {
    printf("Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  printf("Done\n");
  return 0;
}
