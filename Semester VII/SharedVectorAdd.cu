#include <hip/hip_runtime.h>

#include <stdio.h>

// Completed by Jacob Doiron and Andrew Riehl

#define imin(a, b)  (a < b ? a : b)

const int n = 5;
const int threadsPerBlock = 256;
const int blocksPerGrid = imin(32, (n + threadsPerBlock - 1) / threadsPerBlock);


// Takes in three vectors a,b,c and adds a and b into c
__global__ void vectorAdd(const int *a, const int *b, int *c, const int numElements)
{
  // Used to hold results, until they can be added into c
    __shared__ int cache[threadsPerBlock];
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    int cacheIndex = threadIdx.x;
    int temp = 0;
    while (index < numElements) {
        temp += a[index] + b[index];
        index += blockDim.x * gridDim.x;
    }
    cache[cacheIndex] = temp;
  // Called to make sure that all threads are synced
  // All threads running have to call this before continuing on with the code
    __syncthreads();
    if (cacheIndex == 0) {
        c[blockIdx.x] = cache[0];
    }
}

int main()
{
      // Error code to check return values for CUDA calls
      hipError_t err = hipSuccess;

      // Print the vector length to be used, and compute its size
      size_t size = n * sizeof(int);
      printf("[Vector addition of %d elements]\n", n);

      // Allocate the host input vector A
      int *h_A = (int*) malloc(size);

      // Allocate the host input vector B
      int *h_B = (int*) malloc(size);

      // Allocate the host output vector C
      int *h_C = (int*) malloc(size);

      // Verify that allocations succeeded
      if (h_A == NULL || h_B == NULL || h_C == NULL) {
          fprintf(stderr, "Failed to allocate host vectors!\n");
          exit(EXIT_FAILURE);
      }

      // Initialize the host input vectors
      for (int i = 0; i < n; i++) {
          h_A[i] = rand() / 100000;
          h_B[i] = rand() / 100000;
          printf("h1[%d]: %5d, h2[%d]: %5d\n", i, h_A[i], i, h_B[i]);
      }

      // Allocate the device input vector A
      int *d_A = NULL;
      err = hipMalloc((void**) &d_A, size);

      if (err != hipSuccess) {
          fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
          exit(EXIT_FAILURE);
      }

      // Allocate the device input vector B
      int *d_B = NULL;
      err = hipMalloc((void**) &d_B, size);

      if (err != hipSuccess) {
          fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
          exit(EXIT_FAILURE);
      }

      // Allocate the device output vector C
      int *d_C = NULL;
      err = hipMalloc((void**) &d_C, size);

      if (err != hipSuccess) {
          fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
          exit(EXIT_FAILURE);
      }

      // Copy the host input vectors A and B in host memory to the device input vectors in
      // device memory
      printf("Copy input data from the host memory to the CUDA device\n");
      err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

      if (err != hipSuccess) {
          fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
          exit(EXIT_FAILURE);
      }

      err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

      if (err != hipSuccess) {
          fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
          exit(EXIT_FAILURE);
      }

      // Launch the Vector Add CUDA Kernel
      printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
      vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, n);
      err = hipGetLastError();

      if (err != hipSuccess) {
          fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
          exit(EXIT_FAILURE);
      }

      // Copy the device result vector in device memory to the host result vector
      // in host memory.
      printf("Copy output data from the CUDA device to the host memory\n");
      err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

      if (err != hipSuccess) {
          fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
          exit(EXIT_FAILURE);
      }

      for (int i = 0; i < n; i++) {
          printf("h3[%d]: %5d\n", i, h_C[i]);
      }

      // Free device global memory
      err = hipFree(d_A);

      if (err != hipSuccess) {
          fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
          exit(EXIT_FAILURE);
      }

      err = hipFree(d_B);

      if (err != hipSuccess) {
          fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
          exit(EXIT_FAILURE);
      }

      err = hipFree(d_C);

      if (err != hipSuccess) {
          fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
          exit(EXIT_FAILURE);
      }

      // Free host memory
      free(h_A);
      free(h_B);
      free(h_C);

      // Reset the device and exit
      // cudaDeviceReset causes the driver to clean up all state. While
      // not mandatory in normal operation, it is good practice.  It is also
      // needed to ensure correct operation when the application is being
      // profiled. Calling cudaDeviceReset causes all profile data to be
      // flushed before the application exits
      err = hipDeviceReset();

      if (err != hipSuccess) {
          fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
          exit(EXIT_FAILURE);
      }

      printf("Done\n");
      return 0;
}
