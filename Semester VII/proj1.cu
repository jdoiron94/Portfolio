#include <hip/hip_runtime.h>

#include <stdio.h>

// Completed by both Jacob Doiron and Andrew Riehl

__global__ void scaleArray(int *array, const int scalar, const int numElements)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	if (index < numElements) {
		array[index] *= scalar;
		index += stride;
	}
}

/**
* Host main routine
*/
int main()
{
	// Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;

	printf("Number of elements in array: ");
	int numElements;
	scanf("%d", &numElements);
	size_t size = numElements * sizeof(int);

	printf("Scalar: ");
	int scalar;
	scanf("%d", &scalar);

	// Allocate the host input vector A
	int *h_A = (int*) malloc(size);

	// Verify that allocations succeeded
	if (h_A == NULL) {
		fprintf(stderr, "Failed to allocate host vectors!\n");
		exit(EXIT_FAILURE);
	}

	// Initialize the host input vector
	for (int i = 0; i < numElements; i++) {
		h_A[i] = rand() / 100000;
		printf("host[%d] = %d\n", i, h_A[i]);
	}

	// Allocate the device input vector A
	int *d_A = NULL;
	err = hipMalloc((void**) &d_A, size);

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Copy the host input vectors A and B in host memory to the device input vectors in
	// device memory
	printf("Copy input data from the host memory to the CUDA device\n");
	err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Launch the Vector Add CUDA Kernel
	int threadsPerBlock = 256;
	int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
	printf("CUDA kernel launch with %d blocks of %d threads", blocksPerGrid, threadsPerBlock);
	scaleArray<<<blocksPerGrid, threadsPerBlock>>>(d_A, scalar, numElements);
	err = hipGetLastError();

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Copy the device result vector in device memory to the host result vector
	// in host memory.
	printf("Copy output data from the CUDA device to the host memory\n");
	err = hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	for (int i = 0; i < numElements; i++) {
		printf("host[%d] = %d\n", i, h_A[i]);
	}

	// Free device global memory
	err = hipFree(d_A);

	if (err != hipSuccess)	{
		fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Free host memory
	free(h_A);

	// Reset the device and exit
	// cudaDeviceReset causes the driver to clean up all state. While
	// not mandatory in normal operation, it is good practice.  It is also
	// needed to ensure correct operation when the application is being
	// profiled. Calling cudaDeviceReset causes all profile data to be
	// flushed before the application exits
	err = hipDeviceReset();

	if (err != hipSuccess)	{
		fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	printf("Done\n");
	return 0;
}
